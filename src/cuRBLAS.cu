#include "hip/hip_runtime.h"
#include "cuRBLAS/cuRBLAS.cuh"

#include <hip/hip_runtime.h>

namespace cuRBLAS {

//int add_one(int x){
//  return x + 1;
//}

    __global__ void reduceSum(const float *input, float *output, int n) {
        extern __shared__ float sharedData[];

        int tid = threadIdx.x;
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        sharedData[tid] = (idx < n) ? input[idx] : 0.0f;
        __syncthreads();

        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            sharedData[tid] += sharedData[tid + s];
            __syncthreads();
        }

        if (tid == 0) {
            output[threadIdx.x] = sharedData[0];
        }
    }

}