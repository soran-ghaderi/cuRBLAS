#include "hip/hip_runtime.h"
#include "curblas/curblas.h"
#include "curblas/curblas_types.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <memory>
#include <cmath>

// Define M_PI if not available
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif


// for the rand generator, I decided to go with the CURAND lib
///**
// * Custom random number generator state for CUDA
// */
//struct curblasRngState {
//    unsigned long long seed;
//    unsigned long long counter;
//
//    __device__ __host__ curblasRngState(unsigned long long s = 1234ULL) : seed(s), counter(0) {}
//
//    // Simple PRNG based on linear congruential generator
//    __device__ __host__ unsigned int next() {
//        counter = (counter * 1103515245ULL + 12345ULL) ^ seed;
//        return (unsigned int)(counter >> 16);
//    }
//
//    // Generate float in [0, 1)
//    __device__ __host__ float nextFloat() {
//        return (float)next() / (float)UINT_MAX;
//    }
//
//    // Generate normal distributed float (Box-Muller transform)
//    __device__ __host__ float nextGaussian() {
//        static bool hasSpare = false;
//        static float spare;
//
//        if (hasSpare) {
//            hasSpare = false;
//            return spare;
//        }
//
//        hasSpare = true;
//        float u = nextFloat();
//        float v = nextFloat();
//
//        // Use device-compatible math functions
//        #ifdef __CUDA_ARCH__
//        float mag = sqrtf(-2.0f * logf(u));
//        spare = mag * cosf(2.0f * (float)M_PI * v);
//        return mag * sinf(2.0f * (float)M_PI * v);
//        #else
//        float mag = sqrt(-2.0f * log(u));
//        spare = mag * cos(2.0f * (float)M_PI * v);
//        return mag * sin(2.0f * (float)M_PI * v);
//        #endif
//    }
//};

__global__ void setup_rng_kernel(hiprandState *state, unsigned long long seed) {
    hiprand_init(seed, threadIdx.x, 0, state);
}


/**
 * Internal curblas context structure
 */
struct curblasContext {
    // CUDA stream for operations
    hipStream_t stream;
    bool ownsStream;
    
    // Custom random number generation
//    curblasRngState* rng;
    hiprandState* rng;
    unsigned long long seed;
    
    // Configuration
    curblasAccuracy_t accuracy;
    curblasSketchType_t defaultSketchType;
    curblasMath_t mathMode;
    
    // Device information
    int deviceId;
    
    // Version information
    int version;
    
    // Constructor
    curblasContext() : 
        stream(nullptr),
        ownsStream(false),
        rng(nullptr),
        seed(1234ULL),
        accuracy(CURBLAS_ACCURACY_MEDIUM),
        defaultSketchType(CURBLAS_SKETCH_AUTO),
        mathMode(CURBLAS_DEFAULT_MATH),
        deviceId(-1),
        version((CURBLAS_VERSION_MAJOR << 16) | (CURBLAS_VERSION_MINOR << 8) | CURBLAS_VERSION_PATCH)
    {}
};

/*
 * ============================================================================
 * curblas Context Management Implementation
 * ============================================================================
 */

curblasStatus_t curblasCreate(curblasHandle_t* handle) {
    if (handle == nullptr) {
        return CURBLAS_STATUS_INVALID_VALUE;
    }
    
    try {
        // Allocate context
        curblasContext* ctx = new curblasContext();
        
        // Check if CUDA is available and there are devices
        int deviceCount = 0;
        hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);
        if (cudaStatus != hipSuccess || deviceCount == 0) {
            // CUDA not available or no devices - create a CPU-only context
            ctx->deviceId = -1;
            ctx->stream = nullptr;
            ctx->ownsStream = false;
            ctx->rng = nullptr;
            *handle = ctx;
            return CURBLAS_STATUS_SUCCESS;
        }
        
        // Get current device
        cudaStatus = hipGetDevice(&ctx->deviceId);
        if (cudaStatus != hipSuccess) {
            delete ctx;
            return CURBLAS_STATUS_NOT_INITIALIZED;
        }
        
        // Create default stream
        cudaStatus = hipStreamCreate(&ctx->stream);
        if (cudaStatus != hipSuccess) {
            delete ctx;
            return CURBLAS_STATUS_ALLOC_FAILED;
        }
        ctx->ownsStream = true;
        
        // Initialize custom random number generator
//        ctx->rng = new curblasRngState(ctx->seed);
//        if (!ctx->rng) {
//            hipStreamDestroy(ctx->stream);
//            delete ctx;
//            return CURBLAS_STATUS_ALLOC_FAILED;
//        }
//
        cudaStatus = hipMalloc(&ctx->rng, sizeof(hiprandState));
        if (cudaStatus != hipSuccess) {
            hipStreamDestroy(ctx->stream);
            delete ctx;
            return CURBLAS_STATUS_ALLOC_FAILED;
        }

        // Initialize the RNG state using our kernel
        setup_rng_kernel<<<1, 1, 0, ctx->stream>>>(ctx->rng, ctx->seed);
        cudaStatus = hipGetLastError(); // Check for kernel launch errors
        if (cudaStatus != hipSuccess) {
            hipFree(ctx->rng);
            hipStreamDestroy(ctx->stream);
            delete ctx;
            return CURBLAS_STATUS_EXECUTION_FAILED;
        }


        *handle = ctx;
        return CURBLAS_STATUS_SUCCESS;
        
    } catch (const std::bad_alloc&) {
        return CURBLAS_STATUS_ALLOC_FAILED;
    } catch (...) {
        return CURBLAS_STATUS_INTERNAL_ERROR;
    }
}

curblasStatus_t curblasDestroy(curblasHandle_t handle) {
    if (handle == nullptr) {
        return CURBLAS_STATUS_INVALID_VALUE;
    }
    
    curblasContext* ctx = static_cast<curblasContext*>(handle);
    
    // Destroy custom random number generator
    if (ctx->rng) {
//        delete ctx->rng;
        hipFree(ctx->rng);

    }
    
    // Destroy stream if we own it
    if (ctx->stream && ctx->ownsStream) {
        hipStreamDestroy(ctx->stream);
    }
    
    // Free context
    delete ctx;
    
    return CURBLAS_STATUS_SUCCESS;
}

curblasStatus_t curblasGetVersion(curblasHandle_t handle, int* version) {
    if (handle == nullptr || version == nullptr) {
        return CURBLAS_STATUS_INVALID_VALUE;
    }
    
    curblasContext* ctx = static_cast<curblasContext*>(handle);
    *version = ctx->version;
    
    return CURBLAS_STATUS_SUCCESS;
}

curblasStatus_t curblasSetStream(curblasHandle_t handle, hipStream_t streamId) {
    if (handle == nullptr) {
        return CURBLAS_STATUS_INVALID_VALUE;
    }
    
    curblasContext* ctx = static_cast<curblasContext*>(handle);
    
    // For CPU-only contexts, just return success (no CUDA operations)
    if (ctx->deviceId == -1) {
        return CURBLAS_STATUS_SUCCESS;
    }
    
    // Destroy old stream if we own it
    if (ctx->stream && ctx->ownsStream) {
        hipStreamDestroy(ctx->stream);
    }
    
    ctx->stream = streamId;
    ctx->ownsStream = false;  // We don't own external streams
    
    // Note: No need to update stream for custom RNG since it's not tied to CUDA streams
    
    return CURBLAS_STATUS_SUCCESS;
}

curblasStatus_t curblasGetStream(curblasHandle_t handle, hipStream_t* streamId) {
    if (handle == nullptr || streamId == nullptr) {
        return CURBLAS_STATUS_INVALID_VALUE;
    }
    
    curblasContext* ctx = static_cast<curblasContext*>(handle);
    
    // For CPU-only contexts, return null stream
    if (ctx->deviceId == -1) {
        *streamId = nullptr;
        return CURBLAS_STATUS_SUCCESS;
    }
    
    *streamId = ctx->stream;
    
    return CURBLAS_STATUS_SUCCESS;
}

/*
 * ============================================================================
 * curblas Configuration Implementation
 * ============================================================================
 */

curblasStatus_t curblasSetAccuracy(curblasHandle_t handle, curblasAccuracy_t accuracy) {
    if (handle == nullptr) {
        return CURBLAS_STATUS_INVALID_VALUE;
    }
    
    if (accuracy < CURBLAS_ACCURACY_HIGH || accuracy > CURBLAS_ACCURACY_CUSTOM) {
        return CURBLAS_STATUS_INVALID_VALUE;
    }
    
    curblasContext* ctx = static_cast<curblasContext*>(handle);
    ctx->accuracy = accuracy;
    
    return CURBLAS_STATUS_SUCCESS;
}

curblasStatus_t curblasGetAccuracy(curblasHandle_t handle, curblasAccuracy_t* accuracy) {
    if (handle == nullptr || accuracy == nullptr) {
        return CURBLAS_STATUS_INVALID_VALUE;
    }
    
    curblasContext* ctx = static_cast<curblasContext*>(handle);
    *accuracy = ctx->accuracy;
    
    return CURBLAS_STATUS_SUCCESS;
}

curblasStatus_t curblasSetSketchType(curblasHandle_t handle, curblasSketchType_t sketchType) {
    if (handle == nullptr) {
        return CURBLAS_STATUS_INVALID_VALUE;
    }
    
    if (sketchType < CURBLAS_SKETCH_GAUSSIAN || sketchType > CURBLAS_SKETCH_AUTO) {
        return CURBLAS_STATUS_INVALID_VALUE;
    }
    
    curblasContext* ctx = static_cast<curblasContext*>(handle);
    ctx->defaultSketchType = sketchType;
    
    return CURBLAS_STATUS_SUCCESS;
}

curblasStatus_t curblasSetRandomSeed(curblasHandle_t handle, unsigned long long seed) {
    if (handle == nullptr) {
        return CURBLAS_STATUS_INVALID_VALUE;
    }
    
    curblasContext* ctx = static_cast<curblasContext*>(handle);
    ctx->seed = seed;
    
    // Update custom random number generator seed
    if (ctx->rng) {
//        ctx->rng->seed = seed;
//        ctx->rng->counter = 0;  // Reset counter when seed changes
        setup_rng_kernel<<<1, 1, 0, ctx->stream>>>(ctx->rng, ctx->seed);

    }
    
    return CURBLAS_STATUS_SUCCESS;
}

curblasStatus_t curblasSetMathMode(curblasHandle_t handle, curblasMath_t mode) {
    if (handle == nullptr) {
        return CURBLAS_STATUS_INVALID_VALUE;
    }
    
    curblasContext* ctx = static_cast<curblasContext*>(handle);
    ctx->mathMode = mode;
    
    return CURBLAS_STATUS_SUCCESS;
}

/*
 * ============================================================================
 * curblas Utility Functions Implementation
 * ============================================================================
 */

const char* curblasGetStatusString(curblasStatus_t status) {
    switch (status) {
        case CURBLAS_STATUS_SUCCESS:          return "CURBLAS_STATUS_SUCCESS";
        case CURBLAS_STATUS_NOT_INITIALIZED:  return "CURBLAS_STATUS_NOT_INITIALIZED";
        case CURBLAS_STATUS_ALLOC_FAILED:     return "CURBLAS_STATUS_ALLOC_FAILED";
        case CURBLAS_STATUS_INVALID_VALUE:    return "CURBLAS_STATUS_INVALID_VALUE";
        case CURBLAS_STATUS_ARCH_MISMATCH:    return "CURBLAS_STATUS_ARCH_MISMATCH";
        case CURBLAS_STATUS_MAPPING_ERROR:    return "CURBLAS_STATUS_MAPPING_ERROR";
        case CURBLAS_STATUS_EXECUTION_FAILED: return "CURBLAS_STATUS_EXECUTION_FAILED";
        case CURBLAS_STATUS_INTERNAL_ERROR:   return "CURBLAS_STATUS_INTERNAL_ERROR";
        case CURBLAS_STATUS_NOT_SUPPORTED:    return "CURBLAS_STATUS_NOT_SUPPORTED";
        case CURBLAS_STATUS_LICENSE_ERROR:    return "CURBLAS_STATUS_LICENSE_ERROR";
        case CURBLAS_STATUS_INSUFFICIENT_WORKSPACE: return "CURBLAS_STATUS_INSUFFICIENT_WORKSPACE";
        default:                              return "Unknown curblas status";
    }
} 