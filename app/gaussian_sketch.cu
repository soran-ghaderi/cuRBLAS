#include "curblas/curblas.cuh"
//#include "curblas/curblas.h"
//#include "curblas/curblas_types.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <iomanip>
#include <vector>



void printMatrix(const std::vector<float>& vec, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << std::fixed << std::setprecision(4) << std::setw(10) << vec[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    int rows = 4;
    int cols = 5;

    int totalElements = rows * cols;
    long long seed = 112345L;
    float scale = 1.0f;

    std::cout << "Generating a" << rows << " x " << cols << " gaussian sketch matrix." << std::endl;

    std::vector<float> h_sketch(totalElements);

    float* d_sketch;
    hipMalloc((void**)&d_sketch, totalElements * sizeof(float));

    int blockSize = 256;
    int numBlocks = (totalElements + blockSize - 1) / blockSize;

    curblas::generateGaussianSketch<<<numBlocks, blockSize>>>(d_sketch, rows, cols, seed, scale);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "kernel launch failed: " << hipGetErrorString(err) << std::endl;
        hipFree(d_sketch);
        return -1;
    }


    hipDeviceSynchronize();

//  bring the data back:
    hipMemcpy(h_sketch.data(), d_sketch, totalElements * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "result:" << std::endl;

    printMatrix(h_sketch, rows, cols);

    hipFree(d_sketch);


}