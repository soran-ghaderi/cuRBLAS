#include "curblas/curblas.cuh"
//#include "curblas/curblas.h"
//#include "curblas/curblas_types.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <iomanip>
#include <vector>



void printMatrix(const std::vector<float>& vec, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << std::fixed << std::setprecision(4) << std::setw(10) << vec[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    int rows = 4000;
    int cols = 5000;

    int totalElements = rows * cols;
    long long seed = 112345L;
    float scale = 1.0f;

    std::cout << "Generating a" << rows << " x " << cols << " gaussian sketch matrix." << std::endl;

    std::vector<float> h_sketch(totalElements);

    float* d_sketch;
    hipMalloc((void**)&d_sketch, totalElements * sizeof(float));

    int blockSize = 256;

    // int totalElements = rows * cols;
    int elementsPerThread = 4; //test
    int totalThreads = (totalElements + elementsPerThread - 1) / elementsPerThread;
    int numBlocks = (totalThreads + blockSize - 1) / blockSize;

    // int numBlocks = (totalElements + blockSize - 1) / blockSize;

    curblas::generateGaussianSketch<<<numBlocks, blockSize>>>(d_sketch, rows, cols, seed, scale);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "kernel launch failed: " << hipGetErrorString(err) << std::endl;
        hipFree(d_sketch);
        return -1;
    }


    hipDeviceSynchronize();
    hipStream_t stream;
    hipStreamCreate(&stream);

//  bring the data back:
    hipMemcpyAsync(h_sketch.data(), d_sketch, totalElements * sizeof(float), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    std::cout << "result:" << rows << 'x' << cols << std::endl;

//    printMatrix(h_sketch, rows, cols);

    hipFree(d_sketch);


}