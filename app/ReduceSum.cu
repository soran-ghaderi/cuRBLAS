#include "curblas/curblas.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_cooperative_groups.h>


int main() {
    // Example usage of the reduceSum kernel
    int N = 1024;
    float h_input[N];

    for (int i = 0; i < N; ++i) {
        h_input[i] = 1.0f;
    }

    float *d_input, *d_output;
    float h_output;


    // Allocate device memory
    hipMalloc((void**)&d_input, N * sizeof(float));
//    hipMalloc((void**)&d_output, sizeof(float));


    // Launch the kernel
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    hipMalloc((void**)&d_output, blockSize * sizeof(float));
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_output, 0, blockSize * sizeof(float));

//    curblas::reduceSum<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(d_input, d_output, N);
    void *args[] = {&d_input, &d_output, &N};
    hipLaunchCooperativeKernel((void*)curblas::reduceSum, dim3(numBlocks), dim3(blockSize), args, blockSize * sizeof(float));


    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
    }

    // Copy the result back to host
    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);


    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Final Sum (from GPU reduction with Cooperative Groups): " << h_output << std::endl;


    std::vector<float> h_partialSums(numBlocks);
    hipMemcpy(h_partialSums.data(), d_output, numBlocks * sizeof(float), hipMemcpyDeviceToHost);

    float finalSum = 0.0f;
    for (int i = 0; i < numBlocks; ++i) {
        finalSum += h_partialSums[i];
        std::cout << "Partial sum from block " << i << ": " << h_partialSums[i] << std::endl;
    }


    std::cout << "Sum: " << finalSum << std::endl;
    // Clean up
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
